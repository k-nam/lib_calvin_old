#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <cstdio>
#include <iostream>
#include "../util/stopwatch.h"

#define BLOCK_SIZE 25

// gridDim = (n / sizeofblock, same, same)
// blockDim = n
__global__ void matrixMultiKernel(int *a, int *b, int *target, size_t size) {
	size_t const blockSizeInMemory = BLOCK_SIZE * BLOCK_SIZE;
	__shared__ int aBlock[blockSizeInMemory];
	__shared__ int bBlock[blockSizeInMemory];
	__shared__ int targetBlock[blockSizeInMemory];

	size_t iBlock = blockIdx.x;
	size_t jBlock = blockIdx.y;
	size_t kBlock = blockIdx.z;

	// Load some data into cache
	size_t i = threadIdx.x;
	for (size_t j = 0; j < BLOCK_SIZE; j++) {
		aBlock[BLOCK_SIZE*i + j] = a[size*(iBlock*BLOCK_SIZE + i) + (jBlock*BLOCK_SIZE + j)];
		bBlock[BLOCK_SIZE*i + j] = b[size*(jBlock*BLOCK_SIZE + i) + (kBlock*BLOCK_SIZE + j)];
		targetBlock[BLOCK_SIZE*i + j] = 0;
	}
	__syncthreads();
	
	for (size_t j = 0; j < BLOCK_SIZE; j++) {
		for (size_t k = 0; k < BLOCK_SIZE; k++) {
			atomicAdd(&targetBlock[BLOCK_SIZE*i + k], 
				aBlock[BLOCK_SIZE*i + j] * bBlock[BLOCK_SIZE*j + k]);
		}
	}
	__syncthreads();

	// Write back to global memory
	for (size_t j = 0; j < BLOCK_SIZE; j++) {
		atomicAdd(&target[size*(iBlock*BLOCK_SIZE + i) + (kBlock*BLOCK_SIZE + j)],
			targetBlock[BLOCK_SIZE*i + j]);
	}
}

// gridDim = (n, n, n)
// blockDim = 1
__global__ void matrixMultiKernel2(int *a, int *b, int *target, size_t size) {
	size_t i = blockIdx.x;
	size_t j = blockIdx.y;
	size_t k = blockIdx.z;

	__shared__ int block_a; 
	__shared__ int block_b; 
	__shared__ int block_target;

	block_a = a[size*i + j];
	block_b = b[size*j + k];
	//__syncthreads();
	block_target = block_a * block_b;

	atomicAdd(&target[size*i + k], block_target);
}

// gridDim = n
// blockDim = n
__global__ void matrixMultiKernel3(int *a, int *b, int *target, size_t size) {
	size_t i = blockIdx.x;
	size_t j = threadIdx.x;
	for (size_t k = 0; k < size; k++) {
		atomicAdd(&target[size*i + k], a[size*i + j] * b[size*j + k]);
	}
}

hipError_t matrixMulti(int *h_a, int *h_b, int *h_target, size_t size) {
	int *d_a;
	int *d_b;
	int *d_target;
	size_t sizeInMemory = size * size * sizeof(int);
	lib_calvin::stopwatch watch;

	hipMalloc((void**)&d_a, sizeInMemory);
	hipMalloc((void**)&d_b, sizeInMemory);
	hipMalloc((void**)&d_target, sizeInMemory);
	
	hipMemcpy(d_a, h_a, sizeInMemory, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeInMemory, hipMemcpyHostToDevice);
	hipMemcpy(d_target, h_target, sizeInMemory, hipMemcpyHostToDevice);

	size_t numBlocks = size / BLOCK_SIZE;
	dim3 grid(numBlocks, numBlocks, numBlocks);
	
	watch.start();
	matrixMultiKernel<<< grid, BLOCK_SIZE >>>(d_a, d_b, d_target, size);
	//matrixMultiKernel2 <<< dim3(size, size, size), 1 >>>(d_a, d_b, d_target, size);
	//matrixMultiKernel3 <<< size, size >>>(d_a, d_b, d_target, size);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching!\n", cudaStatus);
		goto Error;
	}

	watch.stop();
	std::cout << "matrixMulti took: " << watch.read() << " sec. GFLOPS: " <<
		2 * size*size*size / watch.read() / (1000 * 1000 * 1000) << "\n";

	hipMemcpy(h_target, d_target, sizeInMemory, hipMemcpyDeviceToHost);

Error:
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_target);

	return cudaStatus;
}