#include "hip/hip_runtime.h"


__global__ void matrixMultiKernel(double *a, double *b, double *target, size_t size) {
	int i = blockIdx.x;
	int k = threadIdx.x;
	int sum = 0;
	for (size_t j = 0; j < size; j++) {
		sum += a[size*i + j] * b[size*k + j];
	}
	target[size*i + k] += sum;
}

hipError_t matrixMulti(double *h_a, double *h_b, double *h_target, size_t size) {
	double *d_a;
	double *d_b;
	double *d_target;
	size_t sizeInMemory = size * size * sizeof(double);

	hipMalloc((void**)&d_a, sizeInMemory);
	hipMalloc((void**)&d_b, sizeInMemory);
	hipMalloc((void**)&d_target, sizeInMemory);

	hipMemcpy(d_a, h_a, sizeInMemory, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeInMemory, hipMemcpyHostToDevice);
	hipMemcpy(d_target, h_target, sizeInMemory, hipMemcpyHostToDevice);

	dim3 grid(size, size);
	matrixMultiKernel<<<size, size>>>(d_a, d_b, d_target, size);

	hipDeviceSynchronize();

	hipMemcpy(h_target, d_target, sizeInMemory, hipMemcpyDeviceToHost);

	return hipSuccess;
}